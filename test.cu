#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

//#define NUM_BLOCKS 4
#define NUM_THREADS 512

#define DATA_NUM 512

__global__ void kernel(int* gdata)
{
	__shared__ int sdata[DATA_NUM];

	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	sdata[tid] = gdata[tid];
	__syncthreads();

	sdata[tid] *= sdata[0];
}

int main(int argc, char **argv)
{
	int* sdata = (int*)malloc(sizeof(int) * DATA_NUM);
	for(int i = 0;i < DATA_NUM;i++) sdata[i] = i + 1;

	int* gdata = NULL;
	checkCudaErrors( hipMalloc((void**)&gdata, sizeof(int) * DATA_NUM) );
	checkCudaErrors( hipMemcpy(gdata, sdata, sizeof(int) * DATA_NUM, hipMemcpyHostToDevice) );

	kernel<<<DATA_NUM/NUM_THREADS, NUM_THREADS>>>(gdata);

	checkCudaErrors( hipMemcpy(sdata, gdata, sizeof(int) * DATA_NUM, hipMemcpyDeviceToHost) );

	printf("1～%dの総和→%d\n", DATA_NUM, sdata[1]);

	hipFree(gdata);
	free(sdata);
	
	hipDeviceReset();
}
